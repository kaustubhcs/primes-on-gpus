
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;

#define block_size   32
#define pl_end_number 1000000
#define vector_size 1000

__global__ void prime( int *a, int *b, int *c ) {
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

    if (tid < vector_size){
        c[tid] = a[tid] + b[tid];                   // add vectors together                
    }
}



// ********************** MAIN FUNCTION **********************


int main( void ) { 

    
    cout << "Program Start" << endl;

    hipSetDevice(0);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    bool *small_sieve = new bool [pl_end_number];
    int bool_size = sizeof(bool);

    for (unsigned long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    hipEventRecord(start,0);
    
    for (unsigned long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (unsigned long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tPrime Numbers Computation Time on CPU: %.2f ms\n", time);

 //   cout << "Primes till 100\n";

    unsigned long long int small_sieve_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            //cout << i << " ";
            small_sieve_counter++;
            //cout << small_sieve[i] << "    ";
        }
    }
    cout << endl;

    unsigned long long int *prime_list = new unsigned long long int [small_sieve_counter];

    unsigned long long int inner_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    


    // Pointers in GPU memory
    int *dev_il;
    int *dev_pl;

    // Create Input list
    unsigned long long int start_number = pl_end_number+1;

    unsigned long long int il_size = pl_end_number*pl_end_number;

    bool *input_list = new bool [il_size];

    for (unsigned long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_il,  vector_size * bool_size );
    hipMalloc( (void**)&dev_pl,  small_sieve_counter * bool_size );

    // copy the arrays 'a' and 'b' to the GPU
    // cudaMemcpy( dev_a, a, vector_size * sizeof(int),
    //         cudaMemcpyHostToDevice );
    // cudaMemcpy( dev_b, b, vector_size * sizeof(int),
    //         cudaMemcpyHostToDevice );


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    int grid_size = (vector_size-1)/block_size;
    grid_size++;

    hipEventRecord(start,0);
    //prime<<<grid_size,block_size>>>( dev_a, dev_b, dev_c);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
 //   printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    // cudaMemcpy( c_gpu, dev_c, vector_size * sizeof(int), 
    //         cudaMemcpyDeviceToHost );

    // compare the results
    // int error = 0;
    // for (int i = 0; i < vector_size; i++) {
    //     if (c_cpu[i] != c_gpu[i]){
    //         error = 1;
    //         // printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
    //     }
    //     if (error) break; 
    // }

    // if (error == 0){
    //     printf ("Correct result. No errors were found.\n");
    // }

    // free the memory allocated on the GPU
    // cudaFree( dev_a );
    // cudaFree( dev_b );
    // cudaFree( dev_c );

    // free(a);
    // free(b);
    // free(c_cpu);
    // free(c_gpu);

    return 0;
}

